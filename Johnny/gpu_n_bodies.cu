#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

//add this if compiled by visual studio


#define G 6.67e-2f
#define BLOCK_DIM 1024
#define MAX_RANGE 100.0f
#define MASS 10000.0f
#define EPS 1.0f




void initialize(float3* h_s, float3* h_v, float3* h_a, float dt, int num_bodies) {
	int i, j;
	float3 r;
	float r2, inv_r3;
	for (i = 0; i < num_bodies; i++) {
		h_s[i].x = MAX_RANGE * rand() / (float)RAND_MAX;
		h_s[i].y = MAX_RANGE * rand() / (float)RAND_MAX;
		h_s[i].z = MAX_RANGE * rand() / (float)RAND_MAX;
	}

	for (i = 0; i < num_bodies; i++) {
		for (j = 0; j < num_bodies; j++) {
			if (i != j) {
				r.x = h_s[j].x - h_s[i].x;
				r.y = h_s[j].y - h_s[i].y;
				r.z = h_s[j].z - h_s[i].z;
				r2 = r.x*r.x + r.y*r.y + r.z*r.z + EPS;
				inv_r3 = 1.0f / sqrtf(r2*r2*r2);
				h_a[i].x += inv_r3 * r.x*MASS;
				h_a[i].y += inv_r3 * r.y*MASS;
				h_a[i].z += inv_r3 * r.z*MASS;
			}
		}
	}

	for (i = 0; i < num_bodies; i++) {
		h_v[i].x = h_a[i].x*dt*0.5;
		h_v[i].y = h_a[i].y*dt*0.5;
		h_v[i].z = h_a[i].z*dt*0.5;
	}
}

void cpu_func(float3* h_s, float3* h_v, float3* h_a, float dt, int num_bodies, int num_iteration) {
	int i, j,k;
	float3 r;
	float r2, inv_r3,total_t=0;
	FILE* fp = fopen("pos_cpu.txt","w");
	FILE* fp1 = fopen("accel_cpu.txt", "w");
	for (i = 0; i < num_iteration; i++) {
		clock_t t0 = clock();
		for (j = 0; j < num_bodies; j++) {
			h_s[j].x +=  h_v[j].x*dt;
			h_s[j].y +=  h_v[j].y*dt;
			h_s[j].z +=  h_v[j].z*dt;
			h_v[j].x +=  h_a[j].x*dt;
			h_v[j].y +=  h_a[j].y*dt;
			h_v[j].z +=  h_a[j].z*dt;
		}
		for (j = 0; j< num_bodies; j++) {
			h_a[j].x = 0;
			h_a[j].y = 0;
			h_a[j].z = 0;
			for (k = 0; k < num_bodies; k++) {
				if (j != k) {
					r.x = h_s[k].x - h_s[j].x;
					r.y = h_s[k].y - h_s[j].y;
					r.z = h_s[k].z - h_s[j].z;
					r2 = r.x*r.x + r.y*r.y + r.z*r.z + EPS;
					inv_r3 = 1.0f / sqrtf(r2*r2*r2);
					h_a[j].x += inv_r3 * r.x*MASS;
					h_a[j].y += inv_r3 * r.y*MASS;
					h_a[j].z += inv_r3 * r.z*MASS;
					//printf("cpu: ax[%d]=%.6f\n", j, h_a[j].x);
				}
			}	
		}
		clock_t t1 = clock();
		printf("iteration %d, time cost %.6f\n", i, (float)(t1 - t0) / (float)CLOCKS_PER_SEC);
		total_t += (float)(t1 - t0) / (float)CLOCKS_PER_SEC;
		for (j = 0; j < num_bodies; j++) {
			fprintf(fp1, "%.6f %.6f %.6f\n", h_a[j].x, h_a[j].y, h_a[j].z);
			fprintf(fp, "%.6f %.6f %.6f\n", h_s[j].x, h_s[j].y, h_s[j].z);
		}
	}
	printf("total time %.6f, avg time per iteration %.6f\n", total_t, total_t / (float)num_iteration);
	fclose(fp);
	fclose(fp1);
}


/*
__global__ void initialize(float3* s,float3* v,float3* a,int num_bodies){

}*/

__device__ void force_calc(float3 s,int check_idx,int tile, int num_bodies, float3* a) {
	__shared__ float3 shared_s[BLOCK_DIM];
	int i,N=BLOCK_DIM;
	float3 r;
	float r2, inv_r3;
	if ((tile + 1)*BLOCK_DIM > num_bodies) {
		N = num_bodies % BLOCK_DIM;
	}

	for (i = 0; i < N; i++) {
		if (i!=check_idx) {
			r.x = shared_s[i].x - s.x;
			r.y = shared_s[i].y - s.y;
			r.z = shared_s[i].z - s.z;
			r2 = r.x*r.x + r.y*r.y + r.z*r.z + EPS;
			inv_r3 = 1.0f / sqrtf(r2*r2*r2);
			(*a).x += inv_r3 * r.x*MASS;
			(*a).y += inv_r3 * r.y*MASS;
			(*a).z += inv_r3 * r.z*MASS;
			//printf("gpu a.x=%.6f\n", (*a).x);
		}
		else {
			//printf("sggsgsgs\n");
		}
	}
}


__global__ void accel_update(float3* s, float3* a, int num_bodies) {
	__shared__ float3 shared_s[BLOCK_DIM];
	int gidx = blockDim.x*blockIdx.x + threadIdx.x;
	float3 accel = { 0.0f,0.0f,0.0f }, myPos;
	int idx, i, tile = 0;

	if (gidx < num_bodies) {
		myPos = s[gidx];
	
		
	
	for (i = 0; i < num_bodies; i += blockDim.x) {
		idx = tile * blockDim.x + threadIdx.x;
		if (idx <= num_bodies) {
			shared_s[threadIdx.x] = s[idx];
			__syncthreads();
			force_calc(myPos,threadIdx.x,tile,num_bodies,&accel);
			__syncthreads();
			tile++;
		}
	}
	
	a[gidx] = accel;
	}
}

__global__ void pos_update(float3* s, float3* v, float3* a, float dt, int num_bodies) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < num_bodies) {
		s[i].x += v[i].x*dt;
		s[i].y += v[i].y*dt;
		s[i].z += v[i].z*dt;
		v[i].x += a[i].x*dt;
		v[i].y += a[i].y*dt;
		v[i].z += a[i].z*dt;
	}
}

void gpu_func(float3* s, float3* v, float3* a, float3* h_s, float3* h_a, float dt, int num_bodies, int num_iteration) {
	int i, j;
	FILE *fp = fopen("pos.txt", "w");
	FILE *fp1 = fopen("accel.txt", "w");
	float total_t = 0;
	for (i = 0; i < num_iteration; i++) {
		clock_t t0 = clock();
		pos_update <<<(num_bodies + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM >>> (s, v, a, dt, num_bodies);
		hipDeviceSynchronize();
		accel_update <<<(num_bodies + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM >>> (s, a, num_bodies);
		hipDeviceSynchronize();
		clock_t t1 = clock();
		printf("iteration %d, time cost %.6f\n", i, (float)(t1 - t0) /(float)CLOCKS_PER_SEC);
		total_t += (float)(t1 - t0) / (float)CLOCKS_PER_SEC;
		hipMemcpy(h_s, s, num_bodies * sizeof(float3), hipMemcpyDeviceToHost);
		hipMemcpy( h_a, a, num_bodies * sizeof(float3), hipMemcpyDeviceToHost );
		for (j = 0; j < num_bodies; j++) {
			fprintf(fp, "%.6f %.6f %.6f\n", h_s[j].x, h_s[j].y, h_s[j].z);
			fprintf(fp1, "%.6f %.6f %.6f\n", h_a[j].x, h_a[j].y, h_a[j].z);
		}
	}
	printf("total time %.6f, avg time per iteration %.6f\n", total_t, total_t/(float)num_iteration);
	fclose(fp);
	fclose(fp1);
}


int main(int argc, char *argv[]) {
	if (argc < 2) {
		printf("Error: wrong number of args\n");
		exit(0);
	}
	int num_bodies = atoi(argv[1]);
	float dt = 1;
	if (argc > 2) dt = atof(argv[2]);
	int num_iteration = 10;
	if (argc > 3) num_iteration = atoi(argv[3]);

	//host memory allocation
	float3 *h_s, *h_v, *h_a;
		hipError_t err00 = hipHostMalloc((void**)&h_s, num_bodies * sizeof(float3), hipHostMallocDefault);
	if (err00 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err00), __FILE__, __LINE__);
	}
	hipError_t err01 = hipHostMalloc((void**)&h_v, num_bodies * sizeof(float3), hipHostMallocDefault);
	if (err01 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err01), __FILE__, __LINE__);
	}
	hipError_t err02 = hipHostMalloc((void**)&h_a, num_bodies * sizeof(float3), hipHostMallocDefault);
	if (err02 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err02), __FILE__, __LINE__);
	}

	//initialization
	initialize(h_s, h_v, h_a, dt, num_bodies);

	//device memory allocation
	float3 *s, *v, *a;
	hipError_t err10 = hipMalloc((void**)&s, num_bodies * sizeof(float3));
	if (err10 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err10), __FILE__, __LINE__);
	}
	hipError_t err11 = hipMalloc((void**)&v, num_bodies * sizeof(float3));
	if (err11 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err11), __FILE__, __LINE__);
	}
	hipError_t err12 = hipMalloc((void**)&a, num_bodies * sizeof(float3));
	if (err12 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err12), __FILE__, __LINE__);
	}

	//copy initialized data from host to device
	hipError_t err20 = hipMemcpy(s, h_s, num_bodies * sizeof(float3), hipMemcpyHostToDevice);
	if (err20 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err20), __FILE__, __LINE__);
	}
	hipError_t err21 = hipMemcpy(v, h_v, num_bodies * sizeof(float3), hipMemcpyHostToDevice);
	if (err21 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err21), __FILE__, __LINE__);
	}
	hipError_t err22 = hipMemcpy(a, h_a, num_bodies * sizeof(float3), hipMemcpyHostToDevice);
	if (err22 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err22), __FILE__, __LINE__);
	}
	//cpu code
	printf("cpu code is running....\n");
	cpu_func(h_s,h_v,h_a,dt,num_bodies, num_iteration);
	printf("gpu code is running....\n");
	//run gpu code
	gpu_func(s, v, a, h_s, h_a, dt, num_bodies, num_iteration);

	

	hipDeviceReset();
	hipFree(s); hipFree(v); hipFree(a);
	hipHostFree(h_s); hipHostFree(h_v); hipHostFree(h_a);
}