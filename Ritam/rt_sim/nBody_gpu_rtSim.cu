#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <SFML/Graphics.hpp>
#include "aux_functions_gpu.h"
#include "aux_declarations_gpu_rtsim.h"

// main function
int main (const int argc, const char** argv) {
    printf("\n");
    int nBodies = 30000;
    float dt = DT; // time step
    int nIters = 1000, limit_iter = 0;  // simulation iterations
    int iter = 0, i = 0, grid_size = 0, stop = 0;
    double total_time_gpu = 0;
    
    body_pos *h_body_pos, *d_body_pos;
    body_parameters *h_body_par, *d_body_par;
    
    // reading the arguments (argv data)
    switch(argc)
    {
      case 1:
        printf("------------------------------------------------------\n\n");
        printf("Default values considered, nBodies: 30000.\n\n");
        printf("------------------------------------------------------\n\n");
      break;
      case 2:
        nBodies = atoi(argv[1]);
        printf("------------------------------------------------------\n\n");
        printf("Values considered, nBodies: %i.\n\n", nBodies);
        printf("------------------------------------------------------\n\n");
      break;
      case 3:
        nBodies = atoi(argv[1]);
        nIters = atoi(argv[2]);
        limit_iter = 1;
        printf("------------------------------------------------------\n\n");
        printf("Values considered, nBodies: %i, nIters: %i.\n\n", nBodies, nIters);
        printf("------------------------------------------------------\n\n");
      break;
      default:
        printf("ERR: Invalid number of arguments passed.\n"
               "Aborting...\n");
        return -1;
    }

    // initialising the animation window
    sf::ContextSettings settings;
    settings.antialiasingLevel = 8;
    sf::RenderWindow window(sf::VideoMode(X_RES, Y_RES), "N-Body Simulation", sf::Style::Default, settings);

    // setting up the base body shape
    sf::CircleShape shape(SIZE_OF_BODIES);
    shape.setFillColor(sf::Color::Green);
  
    // host side memory allocation
    size_t pos_bytes = nBodies*sizeof(body_pos);
    size_t par_bytes = nBodies*sizeof(body_parameters);
    hipHostMalloc((body_pos **) &h_body_pos, pos_bytes);
    hipHostMalloc((body_parameters **) &h_body_par, par_bytes); 
    
    // Init mass / pos / vel / acc data
    initialize_bodies(h_body_pos, h_body_par, nBodies);             
  
    // dynamically allocating bodies for the graphics
    std::vector <sf::CircleShape> body_graphics;
    for (i = 0; i < nBodies; i++)
        body_graphics.push_back(shape);
    
    for (i = 0; i < nBodies; i++)
        body_graphics[i].setPosition(h_body_pos[i].x, h_body_pos[i].y);

    // Device side memory allocation  
    hipMalloc((body_pos **) &d_body_pos, pos_bytes);
    hipMalloc((body_parameters **) &d_body_par, par_bytes); 
  
    // determining the grid size
    grid_size = (nBodies+BLOCK_SIZE-1)/BLOCK_SIZE;
  
    // initializing the dim3 variables  
    dim3 block( BLOCK_SIZE, 1, 1 ) ; 
    dim3 grid( grid_size, 1, 1);
    
    // starting the iterations
    printf("---------GPU Data---------\n");
    while((window.isOpen()  && !stop)  || 
         ((limit_iter == 1) && (iter < nIters))) {
        
        // print statements
        if(iter%(nIters/3)==0) {
            printf("iter:%i\n",iter);
            printf("MASS 0\t\t\tMASS 1\t\t\tMASS 2\n");
            printf("x:%.04f\t\tx:%.04f\t\tx:%.04f\n",h_body_pos[0].x,h_body_pos[1].x,h_body_pos[2].x);
            printf("y:%.04f\t\ty:%.04f\t\ty:%.04f\n",h_body_pos[0].y,h_body_pos[1].y,h_body_pos[2].y);
            printf("z:%.04f\t\tz:%.04f\t\tz:%.04f\n",h_body_pos[0].z,h_body_pos[1].z,h_body_pos[2].z);
            printf("\n");
        }

        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed)
                window.close();
        }

        double timeStampA = getTimeStamp();
        // memcopy (host -> device)
        hipMemcpy(d_body_pos, h_body_pos, pos_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_body_par, h_body_par, par_bytes, hipMemcpyHostToDevice);

        // kernel call
        nbody_acc_vel<<<grid, block>>>(d_body_pos,d_body_par,dt,nBodies);
        hipDeviceSynchronize();

        nbody_integration<<<grid, block>>>(d_body_pos,d_body_par,dt,nBodies);
        hipDeviceSynchronize();

        // memcopy (device -> host)
        hipMemcpy(h_body_pos, d_body_pos, pos_bytes, hipMemcpyDeviceToHost);

        double timeStampB = getTimeStamp();
        gpuErrchk(hipPeekAtLastError());
        
        for (i = 0; i < nBodies; i++)
            body_graphics[i].move(h_body_pos[i].x,h_body_pos[i].y);

        window.clear();

        for (i = 0; i < nBodies; i++)
            window.draw(body_graphics[i]);
        window.display();

        total_time_gpu = total_time_gpu + (timeStampB - timeStampA);
        iter++;

        if ((limit_iter == 1) && (iter == nIters))
            stop = 1;
    }

    printf("\n");
    printf("GPU -- Total Time Taken: %lf\n\n", total_time_gpu);
  
    // free memory
    hipHostFree(h_body_pos);
    hipHostFree(h_body_par);
    hipFree(d_body_pos);
    hipFree(d_body_par); 
    hipDeviceReset();
    return 0;
}
  

void initialize_bodies(body_pos *b_pos, body_parameters *b_par, int n) {
    int i = 0;
    srand(time(0));
    for (i = 0; i < n; i++) {
      b_pos[i].x = ((rand() / (float)RAND_MAX) * (X_RES - 100.0f));
      b_pos[i].y = ((rand() / (float)RAND_MAX) * (Y_RES - 100.0f));
      b_pos[i].z = ((rand() / (float)RAND_MAX) * 500.0f);
      b_par[i].m = MASS;
      b_par[i].ax = 0.0f; 
      b_par[i].ay = 0.0f;
      b_par[i].az = 0.0f; 
      b_par[i].vx = 0.0f;
      b_par[i].vy = 0.0f;
      b_par[i].vz = 0.0f;
    }
}  

__global__ void nbody_acc_vel(body_pos* b_pos, body_parameters* b_par, float dt, int n) {
  
    int idx = threadIdx.x + blockIdx.x*blockDim.x ; 
    int j = 0;
    float dx = 0.0f, 
          dy = 0.0f,
          dz = 0.0f,
          sx = 0.0f, 
          sy = 0.0f,
          sz = 0.0f,  
          distSqr = 0.0f,
          distSqr3 = 0.0f, 
          invDist3 = 0.0f;
    
    for (j = 0; j < n; j++) {
      dx = b_pos[j].x - b_pos[idx].x;
      dy = b_pos[j].y - b_pos[idx].y;
      dz = b_pos[j].z - b_pos[idx].z;
      distSqr = dx*dx + dy*dy + dz*dz + EPS;
      distSqr3 = distSqr * distSqr * distSqr;      
      invDist3 = (G * b_par[j].m)/sqrt(distSqr3);
  
      sx += dx * invDist3; sy += dy * invDist3; sz += dz * invDist3;
    }
  
    // acceleration calculation
    b_par[idx].ax += sx;
    b_par[idx].ay += sy;
    b_par[idx].az += sz;
  
    // velocity calculation
    b_par[idx].vx += b_par[idx].ax * dt;
    b_par[idx].vy += b_par[idx].ay * dt;
    b_par[idx].vz += b_par[idx].az * dt;    
}

__global__ void nbody_integration(body_pos* b_pos, body_parameters* b_par, float dt, int n) {
  
    int idx = threadIdx.x + blockIdx.x*blockDim.x ; 

    // integrate and find the new positions
    b_pos[idx].x = b_par[idx].vx*dt;
    b_pos[idx].y = b_par[idx].vy*dt;
    b_pos[idx].z = b_par[idx].vz*dt;
}