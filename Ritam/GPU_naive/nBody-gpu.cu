#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include "aux_functions_gpu.h"
#include "aux_declarations_gpu.h"

// main function
int main (const int argc, const char** argv) {  
  int nBodies = 30000;
  float dt = DT; // time step
  int nIters = 10;  // simulation iterations
  int i = 0, iter = 0, grid_size = 0;
  double total_time = 0;

  bodyStruct *h_body_ds, *d_body_ds;
  
  // reading the arguments (argv data)
  switch(argc)
  {
    case 1:
      printf("Default values considered, nBodies: 30000, nIters: 10.\n");
    break;
    case 2:
      nBodies = atoi(argv[1]);
      printf("Values considered, nBodies: %i, nIters: 10.\n", nBodies);
    break;
    case 3:
      nBodies = atoi(argv[1]);
      nIters = atoi(argv[2]);
      printf("Values considered, nBodies: %i, nIters: %i.\n", nBodies, nIters);
    break;
    default:
      printf("ERR: Invalid number of arguments passed.\n"
             "Aborting...\n");
      return -1;
    break;
  }


  int bytes = nBodies*sizeof(bodyStruct); // memory allocation
  hipHostMalloc( (bodyStruct **) &h_body_ds, bytes );
  initialize_bodies(h_body_ds, nBodies); // Init mass / pos / vel / acc data 

  printf("%f %f %f %f %f %f %f %f %f %f\n",
        h_body_ds[0].m, h_body_ds[0].x, h_body_ds[0].y, h_body_ds[0].z, h_body_ds[0].ax, h_body_ds[0].ay, h_body_ds[0].az, 
        h_body_ds[0].vx, h_body_ds[0].vy, h_body_ds[0].vz);

  // Device side memory allocation

  hipMalloc( (bodyStruct **) &d_body_ds, bytes ) ; 

  // determining the grid size
  grid_size = ceil (nBodies / BLOCK_SIZE);

  // initializing the dim3 variables

  dim3 block( BLOCK_SIZE, 1, 1 ) ; 
  dim3 grid( grid_size, 1, 1);
  FILE *fp=fopen("pos.txt","w");
  // starting the iterations
  for (iter = 0; iter < nIters; iter++) {
    
    printf("Validation: iter %i\tx:%f y:%f z:%f\n",iter+1, h_body_ds[0].x, h_body_ds[0].y, h_body_ds[0].z);
    double timeStampA = getTimeStamp();
    // memcopy (host -> device)
    hipMemcpy( d_body_ds, h_body_ds, bytes, hipMemcpyHostToDevice  ) ;

    // kernel call
    nbody_calculation<<<grid, block>>>(d_body_ds, dt, nBodies);
    hipDeviceSynchronize();

    // memcopy (device -> host)
    hipMemcpy(h_body_ds, d_body_ds, bytes, hipMemcpyDeviceToHost);
    // integrate and find the new positions
    for (i = 0 ; i < nBodies; i++) { 
      h_body_ds[i].x += h_body_ds[i].vx*dt;
      h_body_ds[i].y += h_body_ds[i].vy*dt;
      h_body_ds[i].z += h_body_ds[i].vz*dt;
    }

    double timeStampD = getTimeStamp();

    for (i = 0 ; i < nBodies; i++) { 
     fprintf(fp,"%.6f %.6f %.6f\n",h_body_ds[i].x, h_body_ds[i].y, h_body_ds[i].z);
    }
    total_time = total_time + (timeStampD - timeStampA);
  }
  fclose(fp);
    // printf statements
  printf("%f %f %f %f %f %f %f %f %f %f\n",
        h_body_ds[0].m, h_body_ds[0].x, h_body_ds[0].y, h_body_ds[0].z, h_body_ds[0].ax, h_body_ds[0].ay, h_body_ds[0].az, 
        h_body_ds[0].vx, h_body_ds[0].vy, h_body_ds[0].vz);
  
  printf("Total interactions: %li\tTotal Time Taken: %lf\n",
  (long)(nBodies*nBodies*nIters),total_time);

  // free memory
  hipHostFree( h_body_ds );
  hipFree( d_body_ds   ) ; 
  hipDeviceReset() ;
  return 0;
}

void initialize_bodies(bodyStruct *b, int n) {
  int i = 0;
  srand(time(0));
  for (i = 0; i < n; i++) {
    b[i].m = MASS;
    b[i].x = 2.0f * ((rand() / (float)RAND_MAX) * 100.0f) - 100.0f;
    b[i].y = 2.0f * ((rand() / (float)RAND_MAX) * 100.0f) - 100.0f;
    b[i].z = 2.0f * ((rand() / (float)RAND_MAX) * 100.0f) - 100.0f;
    b[i].ax = 0.0f; 
    b[i].ay = 0.0f;
    b[i].az = 0.0f; 
    b[i].vx = 0.0f;
    b[i].vy = 0.0f;
    b[i].vz = 0.0f;
  }
}

__global__ void nbody_calculation(bodyStruct* b, float dt, int n) {

  int idx = threadIdx.x + blockIdx.x*blockDim.x ; 
  int j = 0;
  float dx = 0.0f, 
        dy = 0.0f,
        dz = 0.0f,
        sx = 0.0f, 
        sy = 0.0f,
        sz = 0.0f,  
        distSqr = 0.0f,
        distSqr3 = 0.0f, 
        invDist3 = 0.0f;
  
  for (j = 0; j < n; j++) {
    dx = b[j].x - b[idx].x;
    dy = b[j].y - b[idx].y;
    dz = b[j].z - b[idx].z;
    distSqr = dx*dx + dy*dy + dz*dz + EPS;
    distSqr3 = distSqr * distSqr * distSqr;      
    invDist3 = (G * b[j].m)/sqrt(distSqr3);

    sx += dx * invDist3; sy += dy * invDist3; sz += dz * invDist3;
  }

  // acceleration calculation
  b[idx].ax += sx;
  b[idx].ay += sy;
  b[idx].az += sz;

  // velocity calculation
  b[idx].vx += b[idx].ax * dt;
  b[idx].vy += b[idx].ay * dt;
  b[idx].vz += b[idx].az * dt;
}
