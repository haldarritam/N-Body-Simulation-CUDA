#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include "aux_functions_gpu.h"
#include "aux_declarations_gpu_cpu.h"

pthread_mutex_t mutex_tid;

// main function
int main (const int argc, const char** argv) {  
  int nBodies = 30000;
  float dt = DT; // time step
  int nIters = 10;  // simulation iterations
  int i = 0, iter = 0, grid_size = 0;
  double total_time = 0;

  thread_arg cpu_body_ds;
  bodyStruct *h_body_ds, *d_body_ds;
  
  // reading the arguments (argv data)
  switch(argc)
  {
    case 1:
      printf("Default values considered, nBodies: 30000, nIters: 10.\n");
    break;
    case 2:
      nBodies = atoi(argv[1]);
      printf("Values considered, nBodies: %i, nIters: 10.\n", nBodies);
    break;
    case 3:
      nBodies = atoi(argv[1]);
      nIters = atoi(argv[2]);
      printf("Values considered, nBodies: %i, nIters: %i.\n", nBodies, nIters);
    break;
    default:
      printf("ERR: Invalid number of arguments passed.\n"
             "Aborting...\n");
      return -1;
    break;
  }

  // initializing the cpu data structure
  cpu_body_ds.n = nBodies;
  cpu_body_ds.dt = dt;

  // host side memory allocation for the GPU execution
  int bytes = nBodies*sizeof(bodyStruct); // memory allocation
  hipHostMalloc( (bodyStruct **) &h_body_ds, bytes );

  // memory allocation for the CPU execution
  bodyStruct *addr = (bodyStruct*)malloc(bytes);
  cpu_body_ds.buf = addr;

  // initializing the data structures
  initialize_bodies(h_body_ds, addr, nBodies);
  printf("%f %f %f %f %f %f %f %f %f %f\n",
        h_body_ds[0].m, h_body_ds[0].x, h_body_ds[0].y, h_body_ds[0].z, h_body_ds[0].ax, h_body_ds[0].ay, h_body_ds[0].az, 
        h_body_ds[0].vx, h_body_ds[0].vy, h_body_ds[0].vz);

  // memory for pthreads
  int pBytes = nBodies*sizeof(pthread_t);
  pthread_t *threads = (pthread_t*)malloc(pBytes);

  // CPU multithreaded execution
  double timeStampB = getTimeStamp();
  for (iter = 0; iter < nIters; iter++) {
    printf("CPU Validation: iter %i\tx:%f y:%f z:%f\n",iter+1, cpu_body_ds.buf[0].x, cpu_body_ds.buf[0].y, cpu_body_ds.buf[0].z);
    cpu_body_ds.tid = 0;
    for (i = 0; i < nBodies; i++)
      pthread_create(&threads[i], NULL, nbody_calculation_cpu, (void *) &cpu_body_ds);

    for (i = 0; i < nBodies; i++) 
      pthread_join(threads[i], NULL);

    for (i = 0 ; i < nBodies; i++) { // integrate position
      cpu_body_ds.buf[i].x += cpu_body_ds.buf[i].vx*dt;
      cpu_body_ds.buf[i].y += cpu_body_ds.buf[i].vy*dt;
    }

  }
  double timeStampC = getTimeStamp();

  printf("\n\n");
  // Device side memory allocation

  hipMalloc( (bodyStruct **) &d_body_ds, bytes ) ; 

  // determining the grid size
  grid_size = ceil (nBodies / BLOCK_SIZE);

  // initializing the dim3 variables

  dim3 block( BLOCK_SIZE, 1, 1 ) ; 
  dim3 grid( grid_size, 1, 1);
  FILE *fp=fopen("pos.txt","w");
  // starting the iterations
  for (iter = 0; iter < nIters; iter++) {
    
    printf("GPU Validation: iter %i\tx:%f y:%f z:%f\n",iter+1, h_body_ds[0].x, h_body_ds[0].y, h_body_ds[0].z);
    double timeStampA = getTimeStamp();
    // memcopy (host -> device)
    hipMemcpy( d_body_ds, h_body_ds, bytes, hipMemcpyHostToDevice  ) ;

    // kernel call
    nbody_calculation_gpu<<<grid, block>>>(d_body_ds, dt, nBodies);
    hipDeviceSynchronize();

    // memcopy (device -> host)
    hipMemcpy(h_body_ds, d_body_ds, bytes, hipMemcpyDeviceToHost);
    // integrate and find the new positions
    for (i = 0 ; i < nBodies; i++) { 
      h_body_ds[i].x += h_body_ds[i].vx*dt;
      h_body_ds[i].y += h_body_ds[i].vy*dt;
      h_body_ds[i].z += h_body_ds[i].vz*dt;
    }

    double timeStampD = getTimeStamp();

    for (i = 0 ; i < nBodies; i++) { 
     fprintf(fp,"%.6f %.6f %.6f\n",h_body_ds[i].x, h_body_ds[i].y, h_body_ds[i].z);
    }
    total_time = total_time + (timeStampD - timeStampA);
  }
  fclose(fp);
    // printf statements
  printf("%f %f %f %f %f %f %f %f %f %f\n",
        h_body_ds[0].m, h_body_ds[0].x, h_body_ds[0].y, h_body_ds[0].z, h_body_ds[0].ax, h_body_ds[0].ay, h_body_ds[0].az, 
        h_body_ds[0].vx, h_body_ds[0].vy, h_body_ds[0].vz);
  
  printf("Total interactions: %li\tTotal Time Taken: %lf\n",
  (long)(nBodies*nBodies*nIters),total_time);

  // free memory
  free(addr);
  hipHostFree( h_body_ds );
  hipFree( d_body_ds   ) ; 
  hipDeviceReset() ;
  return 0;
}

void initialize_bodies(bodyStruct *b, bodyStruct *c, int n) {
  int i = 0;
  srand(time(0));
  for (i = 0; i < n; i++) {
    b[i].m = MASS;
    b[i].x = 2.0f * ((rand() / (float)RAND_MAX) * 100.0f) - 100.0f;
    b[i].y = 2.0f * ((rand() / (float)RAND_MAX) * 100.0f) - 100.0f;
    b[i].z = 2.0f * ((rand() / (float)RAND_MAX) * 100.0f) - 100.0f;
    b[i].ax = 0.0f; 
    b[i].ay = 0.0f;
    b[i].az = 0.0f; 
    b[i].vx = 0.0f;
    b[i].vy = 0.0f;
    b[i].vz = 0.0f;
    
    c[i].m =  b[i].m;
    c[i].x =  b[i].x;  
    c[i].y =  b[i].y;  
    c[i].z =  b[i].z;
    c[i].ax = b[i].ax;
    c[i].ay = b[i].ay;
    c[i].az = b[i].az;
    c[i].vx = b[i].vx;
    c[i].vy = b[i].vy;
    c[i].vz = b[i].vz;
  }
}

void* nbody_calculation_cpu(void* arg) {
  
  thread_arg* b = (thread_arg*) arg;

  pthread_mutex_lock(&mutex_tid);
    int i = b->tid;
    b->tid++;
  pthread_mutex_unlock(&mutex_tid); 
  
  int j = 0;
  float dx = 0.0f, 
        dy = 0.0f,
        dz = 0.0f,
        sx = 0.0f,
        sy = 0.0f,
        sz = 0.0f, 
        distSqr = 0.0f,
        distSqr3 = 0.0f, 
        invDist3 = 0.0f;
  
  for (j = 0; j < b->n; j++) {
    dx = b->buf[j].x - b->buf[i].x;
    dy = b->buf[j].y - b->buf[i].y;
    dz = b->buf[j].z - b->buf[i].z;
    distSqr = dx*dx + dy*dy + dz*dz + EPS;
    distSqr3 = distSqr * distSqr * distSqr;      
    invDist3 = (G * b->buf[j].m)/sqrt(distSqr3);

    sx += dx * invDist3; sy += dy * invDist3; sz += dz * invDist3;
  }

  // acceleration calculation
  b->buf[i].ax += sx;
  b->buf[i].ay += sy;
  b->buf[i].az += sz;

  // velocity calculation
  b->buf[i].vx += b->buf[i].ax * b->dt;
  b->buf[i].vy += b->buf[i].ay * b->dt;
  b->buf[i].vz += b->buf[i].az * b->dt;

  return NULL;
}

__global__ void nbody_calculation_gpu(bodyStruct* b, float dt, int n) {

  int idx = threadIdx.x + blockIdx.x*blockDim.x ; 
  int j = 0;
  float dx = 0.0f, 
        dy = 0.0f,
        dz = 0.0f,
        sx = 0.0f, 
        sy = 0.0f,
        sz = 0.0f,  
        distSqr = 0.0f,
        distSqr3 = 0.0f, 
        invDist3 = 0.0f;
  
  for (j = 0; j < n; j++) {
    dx = b[j].x - b[idx].x;
    dy = b[j].y - b[idx].y;
    dz = b[j].z - b[idx].z;
    distSqr = dx*dx + dy*dy + dz*dz + EPS;
    distSqr3 = distSqr * distSqr * distSqr;      
    invDist3 = (G * b[j].m)/sqrt(distSqr3);

    sx += dx * invDist3; sy += dy * invDist3; sz += dz * invDist3;
  }

  // acceleration calculation
  b[idx].ax += sx;
  b[idx].ay += sy;
  b[idx].az += sz;

  // velocity calculation
  b[idx].vx += b[idx].ax * dt;
  b[idx].vy += b[idx].ay * dt;
  b[idx].vz += b[idx].az * dt;
}
