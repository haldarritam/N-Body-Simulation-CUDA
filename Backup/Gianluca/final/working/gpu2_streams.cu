#include "hip/hip_runtime.h"
#include "nbody_helper2.h"

int main (int argc, char *argv[])
{
	if (argc > 4) {
		printf("Error: Wrong number of arguments.\n");
		exit(EXIT_FAILURE);
	}

	unsigned int nElem = 32768;
	unsigned int nIter = 100;
	unsigned int config = 0;
	char *ptr1, *ptr2, *ptr3;

	// acquiring command line arguments, if any.
	if (argc > 1)	// no. of elements
		nElem  = (unsigned int) strtoul(argv[1], &ptr1, 10);
	if (argc > 2)	// no. of iterations
		nIter  = (unsigned int) strtoul(argv[2], &ptr2, 10);
	if (argc > 3)	// initial config of bodies
		config = (unsigned int) strtoul(argv[3], &ptr3, 10);


	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// SETTING UP DEVICE
	/////////////////////////////////////////////////////////////////////////////////////////////////

	int dev = 0, driverVersion = 0, runtimeVersion = 0;
	hipDeviceProp_t deviceProp;
	checkCudaErrors (hipGetDeviceProperties (&deviceProp, dev));
	checkCudaErrors (hipSetDevice (dev));
	checkCudaErrors (hipDriverGetVersion (&driverVersion));
	checkCudaErrors (hipRuntimeGetVersion (&runtimeVersion));

	print_deviceProperties (dev, driverVersion, runtimeVersion, deviceProp);
	print_simulationParameters (nElem, nIter, NUM_CPU_THREADS);

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// INITIALIZING SIMULATION
	/////////////////////////////////////////////////////////////////////////////////////////////////

	float3 *h_r[2], *h_v;
	float3 *d_r[2], *d_v, *d_a;
	
	size_t nBytes = nElem * sizeof(float3);

	// allocating page-locked memory fobattle for wenothr higher communication bandwidth during real-time vis.
	checkCudaErrors (hipHostMalloc ((void**) &h_r[0], nBytes));
	checkCudaErrors (hipHostMalloc ((void**) &h_r[1], nBytes));
	checkCudaErrors (hipHostMalloc ((void**) &h_v,    nBytes));

	checkCudaErrors (hipMalloc ((void**) &(d_r[0]), nBytes));
	checkCudaErrors (hipMalloc ((void**) &(d_r[1]), nBytes));
	checkCudaErrors (hipMalloc ((void**) &(d_v),    nBytes));
	checkCudaErrors (hipMalloc ((void**) &(d_a),    nBytes));
	

	printf("Initializing bodies' positions / velocities on HOST. Time taken: ");
	double time0 = getTimeStamp();
	init_MassPositionVelocity(h_r[0], h_v, nElem, config);
	printf ("%lfs\n", getTimeStamp()-time0);
	//print_BodyStats(h_m, h_r1, h_v1, h_a1);

	// setting shmem and L1 cache config. 
	// 		hipFuncCachePreferNone:	no preference (default)
	//		hipFuncCachePreferShared:	prefer 48kB shared memory and 16kB L1 cache 
	//		hipFuncCachePreferL1:		prefer 48kB L1 cache and 16kB shmem
	//		hipFuncCachePreferEqual:	prefer 32kB L1 cache and 32kB shmem
	hipFuncCache_t cacheConfig = hipFuncCachePreferShared;
	checkCudaErrors (hipDeviceSetCacheConfig (cacheConfig));

	// copying initialized data from host to device
	checkCudaErrors (hipMemcpy (d_r[0], h_r[0], nBytes, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_r[1], h_r[0], nBytes, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_v,    h_v,    nBytes, hipMemcpyHostToDevice));

	// compute initial acceleration of bodies on device
	dim3 block_size (BLOCK_SIZE);
	dim3 grid_size	((nElem + block_size.x-1)/(block_size.x));
	unsigned int nTiles = (nElem + block_size.x-1)/block_size.x;
	printf("Computing initial acceleration on device. Time Taken: ");
	time0 = getTimeStamp();
	initAcceleration <<<grid_size, block_size, 0, 0>>> (d_a, d_r[0], nTiles);
	hipDeviceSynchronize ();
	printf ("%lfs\n", getTimeStamp()-time0);

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// PERFORMING SIMULATION ON DEVICE
	/////////////////////////////////////////////////////////////////////////////////////////////////
	
	// creating 2 streams for compute and for d2h communication
	hipStream_t *streams = (hipStream_t *) malloc(2*sizeof(hipStream_t));
	hipStreamCreate (&streams[0]);	// d2h communication
	hipStreamCreate (&streams[1]);	// compute

	printf("Computing positions on device.\n");
	double timestamp_GPU_start = getTimeStamp();
	for (unsigned iter=0; iter<nIter; iter++) {
		hipMemcpyAsync (h_r[iter%2], d_r[iter%2], nBytes, 
			hipMemcpyDeviceToHost, streams[0]);
		calcIntegration <<<grid_size, block_size, 0, streams[1]>>> (
			d_r[(iter+1)%2],	// pointer to new positions
			d_r[iter%2], 		// pointer to curr positions
			d_v, 				// pointer to curr velocities
			d_a, 				// pointer to curr accelerations
			nElem, 				// number of bodies in simulation
			nTiles);			// number of shared memory sections per block
		
		checkCudaErrors (hipStreamSynchronize (streams[1]));
	}

	double timestamp_GPU_end = getTimeStamp();
	double elapsedTime = timestamp_GPU_end - timestamp_GPU_start;
	printf("Elapsed Time (total): %.6lfs\n", elapsedTime);
	printf("Elapsed Time per Iteration: %.6lfs\n", elapsedTime/nIter);

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// SIMULATION COMPLETE -- (free memory)
	/////////////////////////////////////////////////////////////////////////////////////////////////

	printf("Freeing global and system memory.\n");

	// stream resources are being released
	checkCudaErrors (hipStreamDestroy (streams[0]));
	checkCudaErrors (hipStreamDestroy (streams[1]));

	// free global memory on GPU DRAM
	checkCudaErrors (hipFree (d_r[0]));
	checkCudaErrors (hipFree (d_r[1]));
	checkCudaErrors (hipFree (d_v));
	checkCudaErrors (hipFree (d_a));
	
	// free page-locked ("pinned") memory on system DRAM
	checkCudaErrors (hipHostFree (h_r[0]));
	checkCudaErrors (hipHostFree (h_r[1]));
	checkCudaErrors (hipHostFree (h_v));

	checkCudaErrors (hipDeviceReset());
	printf("Device successfully reset.\n");

	return 0;
}
