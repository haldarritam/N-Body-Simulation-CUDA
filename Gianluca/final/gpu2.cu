#include "hip/hip_runtime.h"
#include "nbody_helper2.h"

int main (int argc, char *argv[])
{
	if (argc > 4) {
		printf("Error: Wrong number of arguments.\n");
		exit(EXIT_FAILURE);
	}

	unsigned long nElem = 16384;
	unsigned long nIter = 100;
	unsigned int config = 0;
	char *ptr1, *ptr2, *ptr3;

	// acquiring command line arguments, if any.
	if (argc > 1)	// no. of elements
		nElem  = (unsigned int) strtoul(argv[1], &ptr1, 10);
	if (argc > 2)	// no. of iterations
		nIter  = (unsigned int) strtoul(argv[2], &ptr2, 10);
	if (argc > 3)	// initial config of bodies
		config = (unsigned int) strtoul(argv[3], &ptr3, 10);


	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// SETTING UP DEVICE
	/////////////////////////////////////////////////////////////////////////////////////////////////

	int dev = 0, driverVersion = 0, runtimeVersion = 0;
	hipDeviceProp_t deviceProp;
	checkCudaErrors (hipGetDeviceProperties (&deviceProp, dev));
	checkCudaErrors (hipSetDevice (dev));
	checkCudaErrors (hipDriverGetVersion (&driverVersion));
	checkCudaErrors (hipRuntimeGetVersion (&runtimeVersion));

	print_deviceProperties (dev, driverVersion, runtimeVersion, deviceProp);
	print_simulationParameters (nElem, nIter, NUM_CPU_THREADS);

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// INITIALIZING SIMULATION
	/////////////////////////////////////////////////////////////////////////////////////////////////

	float3 *h_dref_r, *h_dref_v;
	float3 *d_r[2],   *d_v,      *d_a;
	
	size_t nBytes = nElem * sizeof(float3);

	// allocating page-locked memory for higher communication bandwidth during real-time vis.
	checkCudaErrors (hipHostMalloc ((void**) &h_dref_r, nBytes));
	checkCudaErrors (hipHostMalloc ((void**) &h_dref_v, nBytes));

	checkCudaErrors (hipMalloc ((void**) &(d_r[0]), nBytes));
	checkCudaErrors (hipMalloc ((void**) &(d_r[1]), nBytes));
	checkCudaErrors (hipMalloc ((void**) &(d_v),    nBytes));
	checkCudaErrors (hipMalloc ((void**) &(d_a),    nBytes));


	printf("Initializing bodies' positions / velocities on HOST. Time taken: ");
	double time0 = getTimeStamp();
	init_MassPositionVelocity(h_dref_r, h_dref_v, nElem, config);
	printf ("%lfs\n", getTimeStamp()-time0);
	//print_BodyStats(h_m, h_r1, h_v1, h_a1);

	// setting shmem and L1 cache config. 
	// 		hipFuncCachePreferNone:	no preference (default)
	//		hipFuncCachePreferShared:	prefer 48kB shared memory and 16kB L1 cache
	//		hipFuncCachePreferL1:		prefer 48kB L1 cache and 16kB shmem
	//		hipFuncCachePreferEqual:	prefer 32kB L1 cache and 32kB shmem
	hipFuncCache_t cacheConfig = hipFuncCachePreferNone;
	checkCudaErrors (hipDeviceSetCacheConfig (cacheConfig));

	// copying initialized data from host to device
	checkCudaErrors (hipMemcpy (d_r[0], h_dref_r, nBytes, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_v,    h_dref_v, nBytes, hipMemcpyHostToDevice));

	// compute initial acceleration of bodies on device
	dim3 block_size (1024);
	dim3 grid_size	((nElem + block_size.x-1)/(block_size.x));
	unsigned int nTiles = (nElem + block_size.x-1)/block_size.x;
	printf("Computing initial acceleration on device. Time Taken: ");
	time0 = getTimeStamp();
	initAcceleration <<<grid_size, block_size, 0, 0>>> (d_a, d_r[0], nTiles);
	hipDeviceSynchronize ();
	printf ("%lfs\n", getTimeStamp()-time0);

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// PERFORMING SIMULATION ON DEVICE
	/////////////////////////////////////////////////////////////////////////////////////////////////

	printf("Computing positions on device. Time taken: ");
	double timestamp_GPU_start = getTimeStamp();
	for (unsigned iter=0; iter<nIter; iter++) {
		calcIntegration <<<grid_size, block_size, 0, 0>>> (
			d_r[(iter+1)%2],	// pointer to new positions
			d_r[iter%2], 		// pointer to curr positions
			d_v, 				// pointer to curr velocities
			d_a, 				// pointer to curr accelerations
			nElem, 				// number of bodies in simulation
			nTiles);			// number of shared memory sections per block
		
		hipDeviceSynchronize ();
		// hipMemcpy(h_dref_r, d_r[(iter+1)%2], nBytes, hipMemcpyDeviceToHost);

		// if (iter%1000 == 0)
		// 	print_BodyStats (gref_m, gref_r, gref_v, gref_a);
	}

	double timestamp_GPU_end = getTimeStamp();
	double elapsedTime = timestamp_GPU_end - timestamp_GPU_start;
	printf("%.6lfs\n", elapsedTime);
	printf("Elapsed Time per Iteration: %.6lfs\n", elapsedTime/nIter);

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// SIMULATION COMPLETE -- (free memory)
	/////////////////////////////////////////////////////////////////////////////////////////////////

	printf("Freeing global and system memory.\n");

	// free global memory on GPU DRAM
	checkCudaErrors (hipFree (d_r[0]));
	checkCudaErrors (hipFree (d_r[1]));
	checkCudaErrors (hipFree (d_v));
	checkCudaErrors (hipFree (d_a));
	
	// free page-locked ("pinned") memory on system DRAM
	checkCudaErrors (hipHostFree (h_dref_r));
	checkCudaErrors (hipHostFree (h_dref_v));

	checkCudaErrors (hipDeviceReset());
	
	printf("Device successfully reset.\n");
	printf("\nElapsed Time: %lfs\n", elapsedTime);
	printf("Average timestep simulation duration: %lfs\n", elapsedTime/nIter); 

	return 0;
}
