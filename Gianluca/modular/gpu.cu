#include "hip/hip_runtime.h"
#include "nbody_helper.h"
#include "nbody_helper_cuda.h"

__global__ void bodyBodyInteraction (float3 ai, float *o_r, float *m, unsigned int j, unsigned int i)
{
	float3 r;
	r.x = o_r[ND*j]   - o_r[ND*i];
	r.y = o_r[ND*j+1] - o_r[ND*i+1];
	r.z = o_r[ND*j+2] - o_r[ND*i+2];
	
	float rDistSquared = r.x*r.x + r.y*r.y + r.z*r.z + SOFTENING;
	float MinvDistCubed = m[j] * rsqrtf(rDistSquared*rDistSquared*rDistSquared);
	
	ai.x = r.x * MinvDistCubed;
	ai.y = r.y * MinvDistCubed;
	ai.z = r.z * MinvDistCubed;
}

__global__ void compute_Device (
	float *o_r, float *o_v, float *o_a, 
	const float *i_r, const float *i_v, const float *i_a, 
	const float *m, const unsigned long nElem)
{
	unsigned long tid = blockIdx.x * blockDim.x + threadIdx.x;
	//if (tid == 0)
		//printf("x: %.2f\ty:%.2f\tz:%.2f\n", i_r[0], i_r[1], i_r[2]);

	float ax_ip1 = 0.0, ay_ip1 = 0.0, az_ip1 = 0.0;
	float dx_ip1, dy_ip1, dz_ip1, rDistSquared, invDistCubed;

	if (tid < nElem) {
		// calculating subsequent position of body (one body per thread)
		o_r[ND*tid]   = i_r[ND*tid]   + i_v[ND*tid]*DT   + i_a[ND*tid]*DTSQd2;		// x-position
		o_r[ND*tid+1] = i_r[ND*tid+1] + i_v[ND*tid+1]*DT + i_a[ND*tid+1]*DTSQd2;	// y-position
		o_r[ND*tid+2] = i_r[ND*tid+2] + i_v[ND*tid+2]*DT + i_a[ND*tid+2]*DTSQd2;	// z-position

		// calculating the NEXT iteration's acceleration and velocity
		#pragma unroll 4
		for (unsigned long j=0; j<nElem; j++) {
			dx_ip1 = o_r[ND*j]   - o_r[ND*tid];
			dy_ip1 = o_r[ND*j+1] - o_r[ND*tid+1];
			dz_ip1 = o_r[ND*j+2] - o_r[ND*tid+2];
			rDistSquared = dx_ip1*dx_ip1 + dy_ip1*dy_ip1 + dz_ip1*dz_ip1 + SOFTENING;
			invDistCubed = m[j] * rsqrtf(rDistSquared*rDistSquared*rDistSquared);
			ax_ip1 += dx_ip1 * invDistCubed;
			ay_ip1 += dy_ip1 * invDistCubed;
			az_ip1 += dz_ip1 * invDistCubed;
		}

		o_a[ND*tid]   = G*ax_ip1;	// x-acceleration
		o_a[ND*tid+1] = G*ay_ip1;	// y-acceleration
		o_a[ND*tid+2] = G*az_ip1;	// z-acceleration

		o_v[ND*tid]   = i_v[ND*tid]   + (i_a[ND*tid]   + ax_ip1)*DTd2;	// x-velocity
		o_v[ND*tid+1] = i_v[ND*tid+1] + (i_a[ND*tid+1] + ay_ip1)*DTd2;	// y-velocity
		o_v[ND*tid+2] = i_v[ND*tid+2] + (i_a[ND*tid+2] + az_ip1)*DTd2;	// z-velocity
	}
}


int main (int argc, char *argv[])
{
	if (argc > 3) {
		printf("Error: Wrong number of arguments.\n");
		exit(EXIT_FAILURE);
	}

	unsigned long nElem = 16384;
	unsigned long nIter = 100;
	char *ptr1, *ptr2;

	if (argc > 1)
		nElem = strtoul(argv[1], &ptr1, 10);
	if (argc > 2)
		nIter = strtoul(argv[2], &ptr2, 10);


	////////////////////////////////////////////////////////////////
	/// SETTING UP DEVICE
	////////////////////////////////////////////////////////////////

	int dev = 0, driverVersion = 0, runtimeVersion = 0;
	hipDeviceProp_t deviceProp;
	checkCudaErrors (hipGetDeviceProperties (&deviceProp, dev));
	checkCudaErrors (hipSetDevice (dev));
	checkCudaErrors (hipDriverGetVersion (&driverVersion));
	checkCudaErrors (hipRuntimeGetVersion (&runtimeVersion));

	printf("\n===== Device Properties ======\n\n");
	printf("  Device %d: %s\n", dev, deviceProp.name);
	printf("  CUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
		driverVersion/1000, (driverVersion%100)/10,
		runtimeVersion/1000, (runtimeVersion%100)/10);
	printf("  CUDA Capability Major/Minor version number: %d.%d\n",
		deviceProp.major, deviceProp.minor);
	printf("  Number of SMs: %d\n", deviceProp.multiProcessorCount);
	printf("  Total amount of global memory: %.2f GB (%llu B)\n",
		(float) deviceProp.totalGlobalMem/pow(1024.0,3),
		(unsigned long long) deviceProp.totalGlobalMem);
	printf("  Total amount of constant memory: %4.2f kB\n", 
		deviceProp.totalConstMem/1024.0);
	printf("  Total amount of shared memory per block: %4.2f kB\n",
		deviceProp.sharedMemPerBlock/1024.0);
	printf("  Total number of registers available per block: %d\n",
		deviceProp.regsPerBlock);
	printf("  Warp size: %d\n", deviceProp.warpSize);
	printf("  Maximum number of threads per block: %d\n",
		deviceProp.maxThreadsPerBlock);
	printf("  Maximum number of threads per SM: %d\n",
		deviceProp.maxThreadsPerMultiProcessor);
	printf("  Maximum number of warps per SM: %d\n",
		deviceProp.maxThreadsPerMultiProcessor/32);
	printf("  Maximum size of each block dimension: %d x %d x %d\n",
		deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
		deviceProp.maxThreadsDim[2]);
	printf("  Maximum size of each grid dimension: %d x %d x %d\n",
		deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);
	printf("  Maximum memory pitch: %lu B\n", deviceProp.memPitch);
	printf("  Memory Clock Rate (MHz): %.1f\n", 
		deviceProp.memoryClockRate/1e3);
	printf("  Memory Bus Width (b): %d\n", deviceProp.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %.2f\n\n",
		2.0*deviceProp.memoryClockRate*(deviceProp.memoryBusWidth/8)/1e6);


	printf("\n===== Simulation Parameters =====\n\n");
	printf("  Number of Bodies = %ld\n", nElem);
	printf("  Number of Time Steps = %ld\n", nIter);
	printf("  Number of CPU Threads = %d\n\n", NUM_CPU_THREADS);
	printf("=================================\n\n\n");

	////////////////////////////////////////////////////////////////
	/// INITIALIZING SIMULATION
	////////////////////////////////////////////////////////////////

	float *h_m, *h_r1, *h_r2, *h_v1, *h_v2, *h_a1, *h_a2;	// host data
	float *d_m, *d_r1, *d_r2, *d_v1, *d_v2, *d_a1, *d_a2;	// device data
	float *gref_r, *gref_v, *gref_a;

	size_t nBytes = nElem * sizeof(float);
	h_m  = (float *) malloc(nBytes);
	h_r1 = (float *) malloc(nBytes*ND);
	h_r2 = (float *) malloc(nBytes*ND);
	h_v1 = (float *) malloc(nBytes*ND);
	h_v2 = (float *) malloc(nBytes*ND);
	h_a1 = (float *) malloc(nBytes*ND);
	h_a2 = (float *) malloc(nBytes*ND);

	gref_r = (float *) malloc(nBytes*ND);
	gref_v = (float *) malloc(nBytes*ND);
	gref_a = (float *) malloc(nBytes*ND);

	memset (h_m,  0, nBytes);
	memset (h_r1, 0, nBytes*ND);
	memset (h_r2, 0, nBytes*ND);
	memset (h_v1, 0, nBytes*ND);
	memset (h_v2, 0, nBytes*ND);
	memset (h_a1, 0, nBytes*ND);
	memset (h_a2, 0, nBytes*ND);

	memset (gref_r, 0, nBytes*ND);
	memset (gref_v, 0, nBytes*ND);
	memset (gref_a, 0, nBytes*ND);

	// initialize data on host size and then transfer to device
	US.m = h_m;
	US.r1 = h_r1;
	US.r2 = h_r2;
	US.v1 = h_v1;
	US.v2 = h_v2;
	US.a1 = h_a1;
	US.a2 = h_a2;
	US.nElem = nElem;
	US.nIter = nIter;

	printf("Initializing bodies on HOST. Time taken: ");
	double time0 = getTimeStamp();
	init_MassPositionVelocity();

	// for portability, explicity create threads in a joinable state
	pthread_t threads [NUM_CPU_THREADS];
	pthread_attr_t attr;
	pthread_attr_init (&attr);
	pthread_attr_setdetachstate (&attr, PTHREAD_CREATE_JOINABLE);

	// creating the threads to calculate initial body accelerations on HOST
	unsigned long i;
	int rc;
	void *status;
	for (i=0; i<NUM_CPU_THREADS; i++) {
		rc = pthread_create (&threads[i], &attr, init_Acceleration_SMT, (void *) i);
		if (rc) {
			printf("Error; return code from pthread_create() is %d.\n", rc);
			exit(EXIT_FAILURE);
		}
	}

	// wait on the other threads after initial body accelerations on HOST
	for (i=0; i<NUM_CPU_THREADS; i++) {
		rc = pthread_join (threads[i], &status);
		if (rc) {
			printf("ERROR; return code from pthread_join() is %d.\n", rc);
			exit(EXIT_FAILURE);
		}
	}
	printf ("%lfs\n", getTimeStamp()-time0);
	//print_BodyStats(h_m, h_r1, h_v1, h_a1);

	// allocating space in device global memory for data
	checkCudaErrors (hipMalloc ((void**) &d_m,  nBytes));
	checkCudaErrors (hipMalloc ((void**) &d_r1, nBytes*ND));
	checkCudaErrors (hipMalloc ((void**) &d_r2, nBytes*ND));
	checkCudaErrors (hipMalloc ((void**) &d_v1, nBytes*ND));
	checkCudaErrors (hipMalloc ((void**) &d_v2, nBytes*ND));
	checkCudaErrors (hipMalloc ((void**) &d_a1, nBytes*ND));
	checkCudaErrors (hipMalloc ((void**) &d_a2, nBytes*ND));

	// copying initialized data from host to device
	checkCudaErrors (hipMemcpy (d_m,  h_m,  nBytes,   hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_r1, h_r1, nBytes*ND, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_r2, h_r2, nBytes*ND, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_v1, h_v1, nBytes*ND, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_v2, h_v2, nBytes*ND, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_a1, h_a1, nBytes*ND, hipMemcpyHostToDevice));
	checkCudaErrors (hipMemcpy (d_a2, h_a2, nBytes*ND, hipMemcpyHostToDevice));

	////////////////////////////////////////////////////////////////
	/// PERFORMING SIMULATION ON DEVICE
	////////////////////////////////////////////////////////////////

	dim3 block (1024);
	dim3 grid  ((nElem+block.x-1)/(block.x));

	double timestamp_GPU_start = getTimeStamp();
	for (unsigned long iter=0; iter<nIter; iter++) {
		if (iter % 2 == 0) {
			compute_Device <<<grid, block, 0, 0>>> (d_r2, d_v2, d_a2, d_r1, d_v1, d_a1, d_m, nElem);
			hipDeviceSynchronize ();
			// hipMemcpy(gref_m, d_m, nBytes, hipMemcpyDeviceToHost);
			// hipMemcpy(gref_r, d_r2, nBytes*2, hipMemcpyDeviceToHost);
			// hipMemcpy(gref_v, d_v2, nBytes*2, hipMemcpyDeviceToHost);
			// hipMemcpy(gref_a, d_a2, nBytes*2, hipMemcpyDeviceToHost);

		} else {
			compute_Device <<<grid, block, 0, 0>>> (d_r1, d_v1, d_a1, d_r2, d_v2, d_a2, d_m, nElem);
			hipDeviceSynchronize ();
			// hipMemcpy(gref_m, d_m, nBytes, hipMemcpyDeviceToHost);
			// hipMemcpy(gref_r, d_r1, nBytes*2, hipMemcpyDeviceToHost);
			// hipMemcpy(gref_v, d_v1, nBytes*2, hipMemcpyDeviceToHost);
			// hipMemcpy(gref_a, d_a1, nBytes*2, hipMemcpyDeviceToHost);
		}
		// if (iter%1000 == 0)
		// 	print_BodyStats (gref_m, gref_r, gref_v, gref_a);
	}
	double timestamp_GPU_end = getTimeStamp();
	double elapsedTime = timestamp_GPU_end - timestamp_GPU_start;

	////////////////////////////////////////////////////////////////
	/// SIMULATION COMPLETE
	////////////////////////////////////////////////////////////////

	hipFree (d_m);
	hipFree (d_r1); hipFree (d_r2);
	hipFree (d_v1); hipFree (d_v2);
	hipFree (d_a1); hipFree (d_a2);

	checkCudaErrors (hipDeviceReset());
	printf("Device successfully reset.\n");
	printf("\nElapsed Time: %lfs\n", elapsedTime);
	printf("Average timestep simulation duration: %lfs\n", elapsedTime/nIter); 


	free (h_m);
	free (h_r1); free (h_r2);
	free (h_v1); free (h_v2);
	free (h_a1); free (h_a2);

	free (gref_r);
	free (gref_v);
	free (gref_a);

	pthread_attr_destroy (&attr);
	//pthread_exit(NULL);

	return 0;
}
